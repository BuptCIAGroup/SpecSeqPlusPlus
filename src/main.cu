#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <algorithm>
#include <vector>
#include <fstream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include "specseq.h"
#include <filesystem>

enum class IOFormat {
    Binary,
    Ascii
};
std::string extractFileNameWithoutExtension(const std::string& filePath) {
    size_t lastSlashPos = filePath.find_last_of("/\\");
    std::string fileName = (lastSlashPos == std::string::npos) ? filePath : filePath.substr(lastSlashPos + 1);

    size_t lastDotPos = fileName.find_last_of('.');
    if (lastDotPos == std::string::npos) {
        return fileName; // No extension found
    }
    return fileName.substr(0, lastDotPos);
}


void printGpuInfo() {
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    CheckCudaError(err, "Failed to get device count");

    if (deviceCount == 0) {
        std::cout << "No CUDA-capable devices found." << std::endl;
        return;
    }

    std::cout << "+" << std::string(50, '-') << "+" << std::endl;
    std::cout << "| " << std::left << std::setw(30) << "Property" << "Value" << "              |" << std::endl;
    std::cout << "+" << std::string(50, '-') << "+" << std::endl;

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        err = hipGetDeviceProperties(&deviceProp, device);
        CheckCudaError(err, "Failed to get device properties");

        std::cout << "| Device " << device << ": " << deviceProp.name << std::string(48 - 10 - std::strlen(deviceProp.name), ' ') << " |" << std::endl;
        std::cout << "| " << std::left << std::setw(30) << "  Compute capability" << deviceProp.major << "." << deviceProp.minor << std::string(17 - std::to_string(deviceProp.major).length() - std::to_string(deviceProp.minor).length(), ' ') << " |" << std::endl;
        std::cout << "| " << std::left << std::setw(30) << "  Total global memory" << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::string(15 - std::to_string(deviceProp.totalGlobalMem / (1024 * 1024)).length(), ' ') << " |" << std::endl;
        std::cout << "| " << std::left << std::setw(30) << "  Shared memory per block" << deviceProp.sharedMemPerBlock / 1024 << " KB" << std::string(15 - std::to_string(deviceProp.sharedMemPerBlock / 1024).length(), ' ') << " |" << std::endl;
        std::cout << "| " << std::left << std::setw(30) << "  Registers per block" << deviceProp.regsPerBlock << std::string(18 - std::to_string(deviceProp.regsPerBlock).length(), ' ') << " |" << std::endl;
        std::cout << "| " << std::left << std::setw(30) << "  Warp size" << deviceProp.warpSize << std::string(18 - std::to_string(deviceProp.warpSize).length(), ' ') << " |" << std::endl;
        std::cout << "| " << std::left << std::setw(30) << "  Memory clock rate" << deviceProp.memoryClockRate / 1000 << " MHz" << std::string(14 - std::to_string(deviceProp.memoryClockRate / 1000).length(), ' ') << " |" << std::endl;
        std::cout << "| " << std::left << std::setw(30) << "  Memory bus width" << deviceProp.memoryBusWidth << " bits" << std::string(13 - std::to_string(deviceProp.memoryBusWidth).length(), ' ') << " |" << std::endl;
        std::cout << "| " << std::left << std::setw(30) << "  Number of multiprocessors" << deviceProp.multiProcessorCount << std::string(18 - std::to_string(deviceProp.multiProcessorCount).length(), ' ') << " |" << std::endl;
        std::cout << "+" << std::string(50, '-') << "+" << std::endl;
    }

    hipDeviceProp_t prop;
    for (int i = 0; i < deviceCount; ++i) {
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device " << i << ": " << prop.name << std::endl;
        std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "Max threads per multiprocessor: " << prop.maxThreadsPerMultiProcessor << std::endl;
    }
}

bool processFiles(const std::string& inputFile, IOFormat inputFormat,
    std::vector<std::vector<int64_t>>& hv_csc, std::vector<std::vector<int64_t>>& hv_neighbor, std::vector<int32_t>& hv_dim) {
    printGpuInfo();
    if (inputFormat == IOFormat::Ascii) {
        std::ifstream input_stream(inputFile.c_str());
        if (input_stream.fail()) {
            std::cerr << "Error: Failed to read from input stream." << std::endl;
            return false;
        }
        std::string cur_line;
        int64_t cur_col = -1;
        std::vector<int64_t> temp_col;
        hv_csc.clear();
        hv_neighbor.clear();
        hv_dim.clear();
        while (getline(input_stream, cur_line)) {
            cur_line.erase(cur_line.find_last_not_of(" \t\n\r\f\v") + 1);
            if (!cur_line.empty() && cur_line[0] != '#') {
                cur_col++;
                std::stringstream ss(cur_line);
                int64_t temp_dim;
                ss >> temp_dim;
                int32_t cdim = temp_dim;
                int64_t temp_int64_t;
                temp_col.clear();
                while (ss >> temp_int64_t) {
                    temp_col.push_back(static_cast<int64_t>(temp_int64_t));
                }
                std::sort(temp_col.begin(), temp_col.end());
                temp_col.erase(std::unique(temp_col.begin(), temp_col.end()), temp_col.end());
                hv_csc.push_back(temp_col);
                hv_dim.push_back(cdim);
                if (cdim == 0) {
                    hv_neighbor.push_back(std::vector<int64_t>());
                }
                else if (cdim == 1) {
                    hv_neighbor[temp_col[0]].push_back(temp_col[1]);
                    hv_neighbor[temp_col[1]].push_back(temp_col[0]);
                }
            }
        }
        input_stream.close();
    }
    else if (inputFormat == IOFormat::Binary) {
        std::ifstream input_stream(inputFile.c_str(), std::ios::binary);
        if (input_stream.fail()) {
            std::cerr << "Error: Failed to read from input stream." << std::endl;
            return false;
        }
        hv_csc.clear();
        hv_neighbor.clear();
        hv_dim.clear();
        int64_t num_cols;
        input_stream.read(reinterpret_cast<char*>(&num_cols), sizeof(num_cols));

        for (int64_t i = 0; i < num_cols; ++i) {
            int64_t temp_dim;
            input_stream.read(reinterpret_cast<char*>(&temp_dim), sizeof(temp_dim));
            int32_t cdim = temp_dim;

            int64_t col_size;
            input_stream.read(reinterpret_cast<char*>(&col_size), sizeof(col_size));

            std::vector<int64_t> temp_col(col_size);
            input_stream.read(reinterpret_cast<char*>(temp_col.data()), col_size * sizeof(int64_t));

            std::sort(temp_col.begin(), temp_col.end());
            temp_col.erase(std::unique(temp_col.begin(), temp_col.end()), temp_col.end());
            hv_csc.push_back(temp_col);
            hv_dim.push_back(cdim);
            if (cdim == 0) {
                hv_neighbor.push_back(std::vector<int64_t>());
            }
            else if (cdim == 1) {
                hv_neighbor[temp_col[0]].push_back(temp_col[1]);
                hv_neighbor[temp_col[1]].push_back(temp_col[0]);
            }
        }
        input_stream.close();
    }

    printf("-----------------------------------------------Reading  completed-----------------------------------------------\n");

    return true;
}

void printHelp() {
    std::cout << "Usage: program <input_file> <output_file> <input_format> <output_format> <model> <ss_block_size> <block_size_growth_rate> <level1_capacity> <level2_capacity> [<level3_capacity>]\n";
    std::cout << "input_format: ascii or binary\n";
    std::cout << "output_format: ascii or binary\n";
    std::cout << "model: \n";
    std::cout << "  0 - No optimizations\n";
    std::cout << "  1 - Enable high-dimensional clearing theorem\n";
    std::cout << "  2 - Enable edge collapsing\n";
    std::cout << "  3 - Enable both high-dimensional clearing theorem and edge collapsing\n";
    std::cout << "ss_block_size: Block size for ss (default 102400)\n";
    std::cout << "block_size_growth_rate: Block size growth rate (default 1.05)\n";
    std::cout << "queue_size: maximum size addition (default 65536)\n";
    std::cout << "level1_capacity: Capacity for level 1 array\n";
    std::cout << "level2_capacity: Capacity for level 2 array\n";
    std::cout << "level3_capacity: Capacity for level 3 array (can be omitted)\n";
    std::cout << "Example: ./specseq++ input.txt output.txt ascii ascii 0 102400 1.05 65536 100 4000 10000\n";
}


void writeVectorToFile(const std::vector<std::pair<int64_t, int64_t>>& vec, const std::string& filename, IOFormat format) {
    std::ofstream file;

    if (format == IOFormat::Binary) {
        file.open(filename, std::ios::out | std::ios::binary);
    }
    else {
        file.open(filename, std::ios::out);
    }

    if (!file.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        return;
    }

    if (format == IOFormat::Ascii) {
        // Write size in ASCII format
        file << vec.size() << std::endl;

        // Write each pair in ASCII format
        for (const auto& p : vec) {
            file << p.first << " " << p.second << std::endl;
        }
    }
    else {
        // Write size in binary format
        int64_t size = vec.size();
        file.write(reinterpret_cast<const char*>(&size), sizeof(size));

        // Write each pair in binary format
        for (const auto& p : vec) {
            file.write(reinterpret_cast<const char*>(&p.first), sizeof(p.first));
            file.write(reinterpret_cast<const char*>(&p.second), sizeof(p.second));
        }
    }

    file.close();
    printf("-----------------------------------------------Writing  completed-----------------------------------------------\n");
}

// 打印矩阵函数，用于调试
void printMatrix(const std::vector<std::vector<int64_t>>& matrix) {
    for (const auto& row : matrix) {
        for (const auto& col : row) {
            std::cout << col << " ";
        }
        std::cout << std::endl;
    }
}

void dualizeMatrix(std::vector<std::vector<int64_t>>& hv_csc, std::vector<int32_t>& hv_dim) {
    // 创建一个新的矩阵来存储对偶化后的结果
    int32_t n = hv_csc.size();
    std::vector<std::vector<int64_t>> transposed(n);

    // Iterate over each column in the original matrix
    for (int col = 0; col < n; ++col) {
        for (int rowIndex : hv_csc[col]) {
            // Calculate the new row and column indices for the transposed matrix
            int newRow = n - 1 - col;
            int newCol = n - 1 - rowIndex;

            // Add the new row index to the corresponding column in the transposed matrix
            transposed[newCol].push_back(newRow);
        }
    }

    // Sort each column in the transposed matrix
    for (auto& col : transposed) {
        std::sort(col.begin(), col.end());
    }
    hv_csc = transposed;
    for (int col = 0; col < n && col < n - col - 1; ++col) {
        std::swap(hv_dim[col], hv_dim[n - col - 1]);
    }
    return;
}

int main(int argc, char* argv[]) {
    if (argc == 2 && (std::string(argv[1]) == "-help" || std::string(argv[1]) == "-h")) {
        printHelp();
        return 0;
    }

    if (argc < 11) {
        std::cerr << "Invalid number of arguments.\n";
        printHelp();
        return 1;
    }

    std::string inputFile = argv[1];
    std::string outputFile = argv[2];
    std::string inputFormatStr = argv[3];
    std::string outputFormatStr = argv[4];
    int32_t model = std::stoi(argv[5]);
    int32_t ssBlockSize = std::stoi(argv[6]);
    float blockSizeGrowthRate = std::stof(argv[7]);

    int32_t queuSize = std::stoi(argv[8]);
    int32_t level_capacity[4] = { 0 };
    level_capacity[1] = std::stoi(argv[9]);
    level_capacity[2] = std::stoi(argv[10]);
    if (argc >= 12) level_capacity[3] = std::stoi(argv[11]);
    IOFormat inputFormat, outputFormat;

    if (inputFormatStr == "ascii") {
        inputFormat = IOFormat::Ascii;
    }
    else if (inputFormatStr == "binary") {
        inputFormat = IOFormat::Binary;
    }
    else {
        std::cerr << "Invalid input format. Format should be 'ascii' or 'binary'.\n";
        printHelp();
        return 1;
    }

    if (outputFormatStr == "ascii") {
        outputFormat = IOFormat::Ascii;
    }
    else if (outputFormatStr == "binary") {
        outputFormat = IOFormat::Binary;
    }
    else {
        std::cerr << "Invalid output format. Format should be 'ascii' or 'binary'.\n";
        printHelp();
        return 1;
    }

    std::cout << "------------------------------------------------------BUPT------------------------------------------------------\n";
    std::vector<std::vector<int64_t>> hv_csc;
    std::vector<std::vector<int64_t>> hv_neighbor;
    std::vector<int32_t> hv_dim;
    processFiles(inputFile, inputFormat, hv_csc, hv_neighbor, hv_dim);
    // if (dualize) {
    //     dualizeMatrix(hv_csc, hv_dim);
    //     // printMatrix(hv_csc);
    // }
    SpecSeqPlusPlus alg(hv_csc, hv_dim, hv_neighbor, level_capacity, model, ssBlockSize, blockSizeGrowthRate, queuSize);
    alg.setDataFileName(extractFileNameWithoutExtension(inputFile));
    float InitTime = alg.GPU_Init();

    // float ComputeTime = alg.GPU_Compute_With_Iner_Block_Diff();
    // float ComputeTime = alg.GPU_Compute_With_Iner_Block_Diff_Naive();
    // float ComputeTime = alg.GPU_Compute_With_Intra_Block_Diff();
    // float ComputeTime = alg.GPU_Compute_With_Intra_Block_Diff_Naive();
    float ComputeTime = alg.GPU_Compute();
    // float ComputeTime = alg.GPU_Compute_Naive();

    printf("Initialization Time: %10.6f seconds\n", InitTime);
    printf("Computation    Time: %10.6f seconds\n", ComputeTime);
    printf("Total          Time: %10.6f seconds\n", InitTime + ComputeTime);
    std::vector<std::pair<int64_t, int64_t>> PersistencePairs = alg.GetPersistencePairs();
    writeVectorToFile(PersistencePairs, outputFile, outputFormat);
    return 0;
}
